#include "hip/hip_runtime.h"
#include <iostream>
#include <string.h>
#include "FreeImage.h"
#include <stdio.h>
#include <stdlib.h>

#define WIDTH 1920
#define HEIGHT 1024
#define BPP 24 // Since we're outputting three 8 bit RGB values

using namespace std;

//! Diviser la matrice en 3 sous matrice pour le cache ( au lieu de faire +1 , +2 )
//! Passer de unsigned int à char ( vus que pixels )

__global__ void sobel(unsigned *d_img, unsigned *d_tmp, unsigned width, unsigned height)
{
  int y = blockIdx.x * blockDim.x + threadIdx.x;
  int x = blockIdx.y * blockDim.y + threadIdx.y;
  if (y < height && x < width)
  {
    int idx = ((y * width) + x) * 3;

    if (y == 0 || x == 0 || y == height - 1 || x == width - 1)
    {
      d_img[idx] = 0;
      d_img[idx + 1] = 0;
      d_img[idx + 2] = 0;
    }
    else
    {

      int idv1 = (y * width + (x - 1) * 3);
      int idv2 = (y * width + (x + 1) * 3);
      int idv3 = (((y - 1) * width + x) * 3);
      int idv4 = (((y + 1) * width + x) * 3);
      int idv5 = (((y - 1) * width) + (x - 1)) * 3;
      int idv6 = (((y - 1) * width) + (x + 1)) * 3;
      int idv7 = (((y + 1) * width) + (x - 1)) * 3;
      int idv8 = (((y + 1) * width) + (x + 1)) * 3;

      int gx = -d_tmp[idv6] - d_tmp[idv5] - 2 * d_tmp[idv3] + d_tmp[idv8] + d_tmp[idv7] + 2 * d_tmp[idv4];

      int gy = -d_tmp[idv6] - d_tmp[idv8] - 2 * d_tmp[idv2] + d_tmp[idv5] + d_tmp[idv7] + 2 * d_tmp[idv1];

      int gn = sqrtf(gx * gx + gy * gy);

      d_img[idx + 0] = gn;

      d_img[idx + 1] = gn;

      d_img[idx + 2] = gn;
    }
  }
}

__global__ void gris(unsigned *d_img, unsigned width, unsigned height)
{
  int y = blockIdx.x * blockDim.x + threadIdx.x;
  int x = blockIdx.y * blockDim.y + threadIdx.y;
  if (y < height && x < width)
  {

    int idx = ((y * width) + x) * 3;
    int grey = d_img[idx + 0] * 0.299 + d_img[idx + 1] * 0.587 + d_img[idx + 2] * 0.114;

    d_img[idx + 0] = grey;
    d_img[idx + 1] = grey;
    d_img[idx + 2] = grey;
    // printf("%d %d %f\n",y,x, count);
  }
}
__global__ void flou(unsigned *d_img, unsigned width, unsigned height)
{

  int y = blockIdx.x * blockDim.x + threadIdx.x;
  int x = blockIdx.y * blockDim.y + threadIdx.y;
  if (y < height && x < width)
  {

    int count = 0, count1 = 0, count2 = 0, c = 0;
    int idx = ((y * width) + x) * 3;
    int idv1 = (((y + 1) * width) + x) * 3;
    int idv2 = (((y - 1) * width) + x) * 3;
    int idv3 = ((y * width) + (x + 1)) * 3;
    int idv4 = ((y * width) + (x - 1)) * 3;

    if (x > 0)
    {
      count += d_img[idv3];
      count1 += d_img[idv3 + 1];
      count2 += d_img[idv3 + 2];
      c++;
    }

    if (x < width - 1)
    {
      count += d_img[idv4];
      count1 += d_img[idv4 + 1];
      count2 += d_img[idv4 + 2];
      c++;
    }

    if (y > 0)
    {
      count += d_img[idv1];
      count1 += d_img[idv1 + 1];
      count2 += d_img[idv1 + 2];
      c++;
    }

    if (y < height - 1)
    {
      count += d_img[idv2];
      count1 += d_img[idv2 + 1];
      count2 += d_img[idv2 + 2];
      c++;
    }

    count += d_img[idx];
    count1 += d_img[idx + 1];
    count2 += d_img[idx + 2];

    c++;

    d_img[idx + 0] = count / c;
    d_img[idx + 1] = count1 / c;
    d_img[idx + 2] = count2 / c;
  }
}

__global__ void saturation(unsigned *d_img, unsigned width, unsigned height)
{

  int y = blockIdx.x * blockDim.x + threadIdx.x;
  int x = blockIdx.y * blockDim.y + threadIdx.y;
  if (y < height && x < width)
  {

    int idx = ((y * width) + x) * 3;
    d_img[idx + 2] = 0xFF / 1.5;
  }
}

__global__ void symhorizontal(unsigned *d_img, unsigned *d_tmp, unsigned width, unsigned height)
{

  int y = blockIdx.x * blockDim.x + threadIdx.x;
  int x = blockIdx.y * blockDim.y + threadIdx.y;
  if (y < height && x < width)
  {
    int ida = ((y * width) + x) * 3;
    int idb = ((width * height) - ((y * width) + x)) * 3;
    d_img[ida + 0] = d_tmp[idb + 0];
    d_img[ida + 1] = d_tmp[idb + 1];
    d_img[ida + 2] = d_tmp[idb + 2];
  }
}

int main(int argc, char **argv)
{
  FreeImage_Initialise();
  const char *PathName = "img.jpg";
  const char *PathDest = "new_img.png";
  // load and decode a regular file
  FREE_IMAGE_FORMAT fif = FreeImage_GetFileType(PathName);

  FIBITMAP *bitmap = FreeImage_Load(FIF_JPEG, PathName, 0);

  if (!bitmap)
    exit(1); // WTF?! We can't even allocate images ? Die !

  unsigned width = FreeImage_GetWidth(bitmap);
  unsigned height = FreeImage_GetHeight(bitmap);
  unsigned pitch = FreeImage_GetPitch(bitmap);

  printf("Processing Image of size %d x %d\n", width, height);

  int size = sizeof(unsigned int) * 3 * width * height;

  unsigned int *img = (unsigned int *)malloc(size);
  unsigned int *d_img = (unsigned int *)malloc(size);
  unsigned int *d_tmp = (unsigned int *)malloc(size);

  BYTE *bits = (BYTE *)FreeImage_GetBits(bitmap);
  for (int y = 0; y < height; y++)
  {
    BYTE *pixel = (BYTE *)bits;
    for (int x = 0; x < width; x++)
    {
      int idx = ((y * width) + x) * 3;
      img[idx + 0] = pixel[FI_RGBA_RED];
      img[idx + 1] = pixel[FI_RGBA_GREEN];
      img[idx + 2] = pixel[FI_RGBA_BLUE];
      pixel += 3;
    }
    // next line
    bits += pitch;
  }

  memcpy(d_img, img, size);
  memcpy(d_tmp, img, size);

  unsigned *d_a, *d_b, *d_c;

  hipMalloc((void **)&d_a, size);
  hipMalloc((void **)&d_b, size);
  hipMalloc((void **)&d_c, size);

  hipMemcpy(d_a, d_img, size, hipMemcpyHostToDevice);

  dim3 dimBlock(32, 32, 1);
  dim3 dimGrid((height / 32) + 1, (width / 32) + 1, 1);

  gris<<<dimGrid, dimBlock>>>(d_a, width, height);

  hipError_t cudaerr = hipDeviceSynchronize();

  hipMemcpy(d_img, d_a, size, hipMemcpyDeviceToHost);

  hipDeviceSynchronize();

  if (cudaerr != hipSuccess)
    printf("kernel launch failed with error \"%s\".\n",
           hipGetErrorString(cudaerr));

  hipMemcpy(d_b, d_img, size, hipMemcpyHostToDevice);
  hipMemcpy(d_c, d_tmp, size, hipMemcpyHostToDevice);

  sobel<<<dimGrid, dimBlock>>>(d_b, d_c, width, height);

  hipMemcpy(d_img, d_b, size, hipMemcpyDeviceToHost);

  cudaerr = hipDeviceSynchronize();
  if (cudaerr != hipSuccess)
    printf("kernel launch failed with error \"%s\".\n",
           hipGetErrorString(cudaerr));

  FIBITMAP *split = FreeImage_Rescale(bitmap, width / 2, height / 2, FILTER_BOX);

  // Copy back
  memcpy(img, d_img, size);

  bits = (BYTE *)FreeImage_GetBits(bitmap);
  for (int y = 0; y < height; y++)
  {
    BYTE *pixel = (BYTE *)bits;
    for (int x = 0; x < width; x++)
    {
      RGBQUAD newcolor;

      int idx = ((y * width) + x) * 3;
      newcolor.rgbRed = img[idx + 0];
      newcolor.rgbGreen = img[idx + 1];
      newcolor.rgbBlue = img[idx + 2];

      if (!FreeImage_SetPixelColor(bitmap, x, y, &newcolor))
      {
        fprintf(stderr, "(%d, %d) Fail...\n", x, y);
      }

      pixel += 3;
    }
    // next line
    bits += pitch;
  }

  if (FreeImage_Save(FIF_PNG, bitmap, PathDest, 0))
    cout << "Image successfully saved ! " << endl;
  FreeImage_DeInitialise(); // Cleanup !

  free(img);
  free(d_img);
  free(d_tmp);
}
