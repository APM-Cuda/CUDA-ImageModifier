#include "hip/hip_runtime.h"
#include <iostream>
#include <string.h>
#include "FreeImage.h"
#include <stdio.h>
#include <stdlib.h>

#define WIDTH 1920
#define HEIGHT 1024
#define BPP 24 // Since we're outputting three 8 bit RGB values

using namespace std;

//! Diviser la matrice en 3 sous matrice pour le cache ( au lieu de faire +1 , +2 )
//! Passer de unsigned int à char ( vus que pixels )

__global__ void sobel(unsigned *d_img, unsigned *d_tmp, unsigned width, unsigned height)
{
  int y = blockIdx.x * blockDim.x + threadIdx.x;
  int x = blockIdx.y * blockDim.y + threadIdx.y;
  if (y < height && x < width)
  {
    int idx = ((y * width) + x) * 3;

    if (y == 0 || x == 0 || y == height - 1 || x == width - 1)
    {
      d_img[idx] = 0;
      d_img[idx + 1] = 0;
      d_img[idx + 2] = 0;
    }
    else
    {

      int idv1 = (y * width + (x - 1) * 3);
      int idv2 = (y * width + (x + 1) * 3);
      int idv3 = (((y - 1) * width + x) * 3);
      int idv4 = (((y + 1) * width + x) * 3);
      int idv5 = (((y - 1) * width) + (x - 1)) * 3;
      int idv6 = (((y - 1) * width) + (x + 1)) * 3;
      int idv7 = (((y + 1) * width) + (x - 1)) * 3;
      int idv8 = (((y + 1) * width) + (x + 1)) * 3;

      int gx = -d_tmp[idv6] - d_tmp[idv5] - 2 * d_tmp[idv3] + d_tmp[idv8] + d_tmp[idv7] + 2 * d_tmp[idv4];

      int gy = -d_tmp[idv6] - d_tmp[idv8] - 2 * d_tmp[idv2] + d_tmp[idv5] + d_tmp[idv7] + 2 * d_tmp[idv1];

      int gn = (gx * gx + gy * gy) / 10000 * (5000);

      d_img[idx + 0] = gn;

      d_img[idx + 1] = gn;

      d_img[idx + 2] = gn;
    }
  }
}

__global__ void canny(unsigned *d_img, unsigned *d_tmp, unsigned width, unsigned height)
{
  int y = blockIdx.x * blockDim.x + threadIdx.x;
  int x = blockIdx.y * blockDim.y + threadIdx.y;
  if (y < height && x < width)
  {
    int idx = ((y * width) + x) * 3;

    if (y == 0 || x == 0 || y == height - 1 || x == width - 1)
    {
      d_img[idx] = 0;
      d_img[idx + 1] = 0;
      d_img[idx + 2] = 0;
      continue;
    }

    int idv1 = (y * width + (x - 1) * 3);
    int idv2 = (y * width + (x + 1) * 3);
    int idv3 = (((y - 1) * width + x) * 3);
    int idv4 = (((y + 1) * width + x) * 3);

    int gx = -d_tmp[idv1] + d_tmp[idv2];

    int gy = -d_tmp[idv3] + d_tmp[idv4];

    int gn = sqrt(gx * gx + gy * gy);

    d_img[idx + 0] = gn;

    d_img[idx + 1] = gn;

    d_img[idx + 2] = gn;
  }
}

__global__ void resize(unsigned *d_img, unsigned width, unsigned height, unsigned newWidth, unsigned newHeight)
{
  int y = blockIdx.x * blockDim.x + threadIdx.x;
  int x = blockIdx.y * blockDim.y + threadIdx.y;
  if (y < height && x < width)
  {
    int idx = ((y * newWidth) + x) * 3;

    float u = (float)x / (float)newWidth * (float)width;
    float v = (float)y / (float)newHeight * (float)height;

    int x1 = (int)u;
    int y1 = (int)v;
    int x2 = x1 + 1;
    int y2 = y1 + 1;

    if (x2 >= width)
    {
      x2 = width - 1;
    }
    if (y2 >= height)
    {
      y2 = height - 1;
    }

    float a = u - (float)x1;
    float b = v - (float)y1;

    int idv1 = (((y1 * width) + x1) * 3);
    int idv2 = (((y1 * width) + x2) * 3);
    int idv3 = (((y2 * width) + x1) * 3);
    int idv4 = (((y2 * width) + x2) * 3);

    img_new[idx] = (BYTE)(img[idv1] * (1.0f - a) * (1.0f - b) + img[idv2] * a * (1.0f - b) + img[idv3] * (1.0f - a) * b + img[idv4] * a * b);
    img_new[idx + 1] = (BYTE)(img[idv1 + 1] * (1.0f - a) * (1.0f - b) + img[idv2 + 1] * a * (1.0f - b) + img[idv3 + 1] * (1.0f - a) * b + img[idv4 + 1] * a * b);
    img_new[idx + 2] = (BYTE)(img[idv1 + 2] * (1.0f - a) * (1.0f - b) + img[idv2 + 2] * a * (1.0f - b) + img[idv3 + 2] * (1.0f - a) * b + img[idv4 + 2] * a * b);
  }
}

__global__ void onlyRouge(unsigned *d_img, unsigned width, unsigned height)
{
  int y = blockIdx.x * blockDim.x + threadIdx.x;
  int x = blockIdx.y * blockDim.y + threadIdx.y;
  if (y < height && x < width)
  {

    int idx = ((y * width) + x) * 3;

    d_img[idx + 1] = 0;
    d_img[idx + 2] = 0;
  }
}

__global__ void onlyBleu(unsigned *d_img, unsigned width, unsigned height)
{
  int y = blockIdx.x * blockDim.x + threadIdx.x;
  int x = blockIdx.y * blockDim.y + threadIdx.y;
  if (y < height && x < width)
  {

    int idx = ((y * width) + x) * 3;

    d_img[idx] = 0;
    d_img[idx + 1] = 0;
  }
}

__global__ void onlyVert(unsigned *d_img, unsigned width, unsigned height)
{
  int y = blockIdx.x * blockDim.x + threadIdx.x;
  int x = blockIdx.y * blockDim.y + threadIdx.y;
  if (y < height && x < width)
  {

    int idx = ((y * width) + x) * 3;

    d_img[idx] = 0;
    d_img[idx + 2] = 0;
  }
}

__global__ void diapositive(unsigned *d_img, unsigned width, unsigned height)
{
  int y = blockIdx.x * blockDim.x + threadIdx.x;
  int x = blockIdx.y * blockDim.y + threadIdx.y;
  if (y < height && x < width)
  {

    int idx = ((y * width) + x) * 3;

    d_img[idx + 0] = 255 - d_img[idx];
    d_img[idx + 1] = 255 - d_img[idx + 1];
    d_img[idx + 2] = 255 - d_img[idx + 2];
  }
}

__global__ void gris(unsigned *d_img, unsigned width, unsigned height)
{
  int y = blockIdx.x * blockDim.x + threadIdx.x;
  int x = blockIdx.y * blockDim.y + threadIdx.y;
  if (y < height && x < width)
  {

    int idx = ((y * width) + x) * 3;
    int grey = d_img[idx + 0] * 0.299 + d_img[idx + 1] * 0.587 + d_img[idx + 2] * 0.114;

    d_img[idx + 0] = grey;
    d_img[idx + 1] = grey;
    d_img[idx + 2] = grey;
  }
}
__global__ void flou(unsigned *d_img, unsigned width, unsigned height)
{

  int y = blockIdx.x * blockDim.x + threadIdx.x;
  int x = blockIdx.y * blockDim.y + threadIdx.y;
  if (y < height && x < width)
  {

    int count = 0, count1 = 0, count2 = 0, c = 0;
    int idx = ((y * width) + x) * 3;
    int idv1 = (((y + 1) * width) + x) * 3;
    int idv2 = (((y - 1) * width) + x) * 3;
    int idv3 = ((y * width) + (x + 1)) * 3;
    int idv4 = ((y * width) + (x - 1)) * 3;

    if (x > 0)
    {
      count += d_img[idv3];
      count1 += d_img[idv3 + 1];
      count2 += d_img[idv3 + 2];
      c++;
    }

    if (x < width - 1)
    {
      count += d_img[idv4];
      count1 += d_img[idv4 + 1];
      count2 += d_img[idv4 + 2];
      c++;
    }

    if (y > 0)
    {
      count += d_img[idv1];
      count1 += d_img[idv1 + 1];
      count2 += d_img[idv1 + 2];
      c++;
    }

    if (y < height - 1)
    {
      count += d_img[idv2];
      count1 += d_img[idv2 + 1];
      count2 += d_img[idv2 + 2];
      c++;
    }

    count += d_img[idx];
    count1 += d_img[idx + 1];
    count2 += d_img[idx + 2];

    c++;

    d_img[idx + 0] = count / c;
    d_img[idx + 1] = count1 / c;
    d_img[idx + 2] = count2 / c;
  }
}

__global__ void saturationBleu(unsigned *d_img, unsigned width, unsigned height)
{

  int y = blockIdx.x * blockDim.x + threadIdx.x;
  int x = blockIdx.y * blockDim.y + threadIdx.y;
  if (y < height && x < width)
  {

    int idx = ((y * width) + x) * 3;
    d_img[idx + 2] = 0xFF / 1.5;
  }
}

__global__ void saturationRouge(unsigned *d_img, unsigned width, unsigned height)
{

  int y = blockIdx.x * blockDim.x + threadIdx.x;
  int x = blockIdx.y * blockDim.y + threadIdx.y;
  if (y < height && x < width)
  {

    int idx = ((y * width) + x) * 3;
    d_img[idx] = 0xFF / 1.5;
  }
}

__global__ void saturationVert(unsigned *d_img, unsigned width, unsigned height)
{

  int y = blockIdx.x * blockDim.x + threadIdx.x;
  int x = blockIdx.y * blockDim.y + threadIdx.y;
  if (y < height && x < width)
  {

    int idx = ((y * width) + x) * 3;
    d_img[idx + 1] = 0xFF / 1.5;
  }
}

__global__ void saturationCyan(unsigned *d_img, unsigned width, unsigned height)
{

  int y = blockIdx.x * blockDim.x + threadIdx.x;
  int x = blockIdx.y * blockDim.y + threadIdx.y;
  if (y < height && x < width)
  {

    int idx = ((y * width) + x) * 3;
    d_img[idx + 1] = 0xFF / 1.5;
    d_img[idx + 2] = 0xFF / 1.5;
  }
}
__global__ void symhorizontal(unsigned *d_img, unsigned *d_tmp, unsigned width, unsigned height)
{

  int y = blockIdx.x * blockDim.x + threadIdx.x;
  int x = blockIdx.y * blockDim.y + threadIdx.y;
  if (y < height && x < width)
  {
    int ida = ((y * width) + x) * 3;
    int idb = ((width * height) - ((y * width) + x)) * 3;
    d_img[ida + 0] = d_tmp[idb + 0];
    d_img[ida + 1] = d_tmp[idb + 1];
    d_img[ida + 2] = d_tmp[idb + 2];
  }
}

int main(int argc, char **argv)
{
  FreeImage_Initialise();
  const char *PathName = "img.jpg";
  const char *PathDest = "new_img.png";
  // load and decode a regular file
  FREE_IMAGE_FORMAT fif = FreeImage_GetFileType(PathName);

  FIBITMAP *bitmap = FreeImage_Load(FIF_JPEG, PathName, 0);

  if (!bitmap)
    exit(1); // WTF?! We can't even allocate images ? Die !

  unsigned width = FreeImage_GetWidth(bitmap);
  unsigned height = FreeImage_GetHeight(bitmap);
  unsigned pitch = FreeImage_GetPitch(bitmap);

  printf("Processing Image of size %d x %d\n", width, height);

  int size = sizeof(unsigned int) * 3 * width * height;

  unsigned int *img = (unsigned int *)malloc(size);
  unsigned int *d_img = (unsigned int *)malloc(size);
  unsigned int *d_tmp = (unsigned int *)malloc(size);

  BYTE *bits = (BYTE *)FreeImage_GetBits(bitmap);
  for (int y = 0; y < height; y++)
  {
    BYTE *pixel = (BYTE *)bits;
    for (int x = 0; x < width; x++)
    {
      int idx = ((y * width) + x) * 3;
      img[idx + 0] = pixel[FI_RGBA_RED];
      img[idx + 1] = pixel[FI_RGBA_GREEN];
      img[idx + 2] = pixel[FI_RGBA_BLUE];
      pixel += 3;
    }
    // next line
    bits += pitch;
  }

  memcpy(d_img, img, size);
  memcpy(d_tmp, img, size);

  unsigned *d_a, *d_b, *d_c;

  hipMalloc((void **)&d_a, size);
  hipMalloc((void **)&d_b, size);
  hipMalloc((void **)&d_c, size);

  hipMemcpy(d_a, d_img, size, hipMemcpyHostToDevice);

  dim3 dimBlock(32, 32, 1);
  dim3 dimGrid((height / 32) + 1, (width / 32) + 1, 1);

  onlyRouge<<<dimGrid, dimBlock>>>(d_a, width, height);

  // sobel<<<dimGrid, dimBlock>>>(d_b, d_c, width, height);

  hipMemcpy(d_img, d_a, size, hipMemcpyDeviceToHost);

  FIBITMAP *sobel = FreeImage_Load(FIF_JPEG, PathName, 0);

  hipDeviceSynchronize();

  bits = (BYTE *)FreeImage_GetBits(sobel);
  for (int y = 0; y < height; y++)
  {
    BYTE *pixel = (BYTE *)bits;
    for (int x = 0; x < width; x++)
    {
      RGBQUAD newcolor;

      int idx = ((y * width) + x) * 3;
      newcolor.rgbRed = d_img[idx + 0];
      newcolor.rgbGreen = d_img[idx + 1];
      newcolor.rgbBlue = d_img[idx + 2];

      if (!FreeImage_SetPixelColor(sobel, x, y, &newcolor))
      {
        fprintf(stderr, "(%d, %d) Fail...\n", x, y);
      }

      pixel += 3;
    }
    // next line
    bits += pitch;
  }

  // POP-ART

  FIBITMAP *split = FreeImage_Rescale(bitmap, width / 2, height / 2, FILTER_BOX);

  unsigned widthSplt = FreeImage_GetWidth(split);
  unsigned heightSplt = FreeImage_GetHeight(split);
  unsigned pitchSplt = FreeImage_GetPitch(split);

  printf("Processing Image of size %d x %d\n", widthSplt, heightSplt);

  int sizeSplt = sizeof(unsigned int) * 3 * widthSplt * heightSplt;

  unsigned int *imgSplt = (unsigned int *)malloc(sizeSplt);
  unsigned int *d_tmpSplt = (unsigned int *)malloc(sizeSplt);

  BYTE *bitsSplt = (BYTE *)FreeImage_GetBits(split);
  for (int y = 0; y < heightSplt; y++)
  {
    BYTE *pixelSplt = (BYTE *)bitsSplt;
    for (int x = 0; x < widthSplt; x++)
    {
      int idx = ((y * widthSplt) + x) * 3;
      imgSplt[idx + 0] = pixelSplt[FI_RGBA_RED];
      imgSplt[idx + 1] = pixelSplt[FI_RGBA_GREEN];
      imgSplt[idx + 2] = pixelSplt[FI_RGBA_BLUE];
      pixelSplt += 3;
    }
    // next line
    bitsSplt += pitchSplt;
  }

  unsigned int *d_imgSplt;
  unsigned int *d_imgSplt2;
  unsigned int *d_imgSplt3;
  unsigned int *d_imgSplt4;

  hipHostMalloc((void **)&d_imgSplt2, sizeSplt);
  hipHostMalloc((void **)&d_imgSplt3, sizeSplt);
  hipHostMalloc((void **)&d_imgSplt4, sizeSplt);
  hipHostMalloc((void **)&d_imgSplt, sizeSplt);

  memcpy(d_imgSplt, imgSplt, sizeSplt);

  memcpy(d_imgSplt2, imgSplt, sizeSplt);
  memcpy(d_imgSplt3, imgSplt, sizeSplt);
  memcpy(d_imgSplt4, imgSplt, sizeSplt);

  unsigned *d1, *d2, *d3, *d4;

  hipMalloc((void **)&d1, sizeSplt);
  hipMalloc((void **)&d2, sizeSplt);
  hipMalloc((void **)&d3, sizeSplt);
  hipMalloc((void **)&d4, sizeSplt);

  hipStream_t stream[4];

  hipStreamCreate(&stream[0]);
  hipStreamCreate(&stream[1]);
  hipStreamCreate(&stream[2]);
  hipStreamCreate(&stream[3]);

  hipMemcpyAsync(d1, d_imgSplt, sizeSplt, hipMemcpyHostToDevice, stream[0]);
  hipMemcpyAsync(d2, d_imgSplt2, sizeSplt, hipMemcpyHostToDevice, stream[1]);
  hipMemcpyAsync(d3, d_imgSplt3, sizeSplt, hipMemcpyHostToDevice, stream[2]);
  hipMemcpyAsync(d4, d_imgSplt4, sizeSplt, hipMemcpyHostToDevice, stream[3]);

  saturationRouge<<<dimGrid, dimBlock, 0, stream[0]>>>(d1, widthSplt, heightSplt);
  saturationCyan<<<dimGrid, dimBlock, 0, stream[1]>>>(d2, widthSplt, heightSplt);
  saturationBleu<<<dimGrid, dimBlock, 0, stream[2]>>>(d3, widthSplt, heightSplt);
  saturationVert<<<dimGrid, dimBlock, 0, stream[3]>>>(d4, widthSplt, heightSplt);

  hipMemcpyAsync(d_imgSplt, d1, sizeSplt, hipMemcpyDeviceToHost, stream[0]);
  hipMemcpyAsync(d_imgSplt2, d2, sizeSplt, hipMemcpyDeviceToHost, stream[1]);
  hipMemcpyAsync(d_imgSplt3, d3, sizeSplt, hipMemcpyDeviceToHost, stream[2]);
  hipMemcpyAsync(d_imgSplt4, d4, sizeSplt, hipMemcpyDeviceToHost, stream[3]);

  hipStreamSynchronize(stream[0]);
  hipStreamSynchronize(stream[1]);
  hipStreamSynchronize(stream[2]);
  hipStreamSynchronize(stream[3]);

  // Copy back
  // memcpy(img, d_img, size);

  bits = (BYTE *)FreeImage_GetBits(bitmap);
  for (int y = 0; y < heightSplt; y++)
  {
    BYTE *pixel = (BYTE *)bits;
    for (int x = 0; x < widthSplt; x++)
    {
      RGBQUAD newcolor;

      int idx = ((y * widthSplt) + x) * 3;
      newcolor.rgbRed = d_imgSplt[idx + 0];
      newcolor.rgbGreen = d_imgSplt[idx + 1];
      newcolor.rgbBlue = d_imgSplt[idx + 2];

      if (!FreeImage_SetPixelColor(bitmap, x, y, &newcolor))
      {
        fprintf(stderr, "(%d, %d) Fail...\n", x, y);
      }

      pixel += 3;
    }
    // next line
    bits += pitch;
  }

  bitsSplt = (BYTE *)FreeImage_GetBits(bitmap);

  for (int y = 0; y < heightSplt; y++)
  {
    BYTE *pixel = (BYTE *)bitsSplt;
    for (int x = 0; x < widthSplt; x++)
    {
      RGBQUAD newcolor;

      int idx = ((y * widthSplt) + x) * 3;
      newcolor.rgbRed = d_imgSplt2[idx + 0];
      newcolor.rgbGreen = d_imgSplt2[idx + 1];
      newcolor.rgbBlue = d_imgSplt2[idx + 2];

      if (!FreeImage_SetPixelColor(bitmap, x + widthSplt, y + heightSplt, &newcolor))
      {
        fprintf(stderr, "(%d, %d) Fail...\n", x, y);
      }

      pixel += 3;
    }
    // next line
    bitsSplt += pitchSplt;
  }
  bitsSplt = (BYTE *)FreeImage_GetBits(bitmap);

  for (int y = 0; y < heightSplt; y++)
  {
    BYTE *pixel = (BYTE *)bitsSplt;
    for (int x = 0; x < widthSplt; x++)
    {
      RGBQUAD newcolor;

      int idx = ((y * widthSplt) + x) * 3;
      newcolor.rgbRed = d_imgSplt3[idx + 0];
      newcolor.rgbGreen = d_imgSplt3[idx + 1];
      newcolor.rgbBlue = d_imgSplt3[idx + 2];

      if (!FreeImage_SetPixelColor(bitmap, x, y + heightSplt, &newcolor))
      {
        fprintf(stderr, "(%d, %d) Fail...\n", x, y);
      }

      pixel += 3;
    }
    // next line
    bitsSplt += pitchSplt;
  }
  bitsSplt = (BYTE *)FreeImage_GetBits(bitmap);

  for (int y = 0; y < heightSplt; y++)
  {
    BYTE *pixel = (BYTE *)bitsSplt;
    for (int x = 0; x < widthSplt; x++)
    {
      RGBQUAD newcolor;

      int idx = ((y * widthSplt) + x) * 3;
      newcolor.rgbRed = d_imgSplt4[idx + 0];
      newcolor.rgbGreen = d_imgSplt4[idx + 1];
      newcolor.rgbBlue = d_imgSplt4[idx + 2];

      if (!FreeImage_SetPixelColor(bitmap, x + widthSplt, y, &newcolor))
      {
        fprintf(stderr, "(%d, %d) Fail...\n", x, y);
      }

      pixel += 3;
    }
    // next line
    bits += pitchSplt;
  }

  if (FreeImage_Save(FIF_PNG, bitmap, PathDest, 0))
    cout << "Image successfully saved ! " << endl;
  if (FreeImage_Save(FIF_PNG, sobel, "sobel.png", 0))
    cout << "Image successfully saved ! " << endl;
  FreeImage_DeInitialise(); // Cleanup !

  /*  free(img);
    free(d_img);
    free(d_tmp);*/
}
